#include <stdio.h>
#include <hip/hip_runtime.h>



void print_array(float *A, int n, int m)
{
    for(int i=0; i<n; i++)
	{
		for (int j=0; j<m; j++)
		{
			printf("%.1f ", A[i*m+j]);
		}
    printf("\n");
	}
}


__global__ void
process_kernel1(float *input, float *output, int n, int m)
{
    // Code for i
	int i= blockIdx.y * blockDim .y+ threadIdx .y;
	int j= blockIdx.x * blockDim.x+ threadIdx.x;

	if ((i<n) && (j<m)) {
	for(int l=0; l<n; l++){
		for (int k = 0; k < m; k+=2) {
			output[i*l+k] = input[i*l+k+1];
			output[i*l+k+1] = input[i*l+k];
			}
	}
	}
}


int main(void)
{
    hipError_t err = hipSuccess;

	int test_cases;
    scanf("%d",&test_cases);
	
	int m, n;
	scanf("%d %d", &m, &n);
	
    size_t size = m*n*sizeof(float);

    float *h_input = (float *)malloc(size);
	float *h_output = (float *)malloc(size);

    if (h_input == NULL || h_output == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < n*m; ++i)
    {
        scanf("%f",&h_input[i]);
        
    }

    float *d_input = NULL;
    err = hipMalloc((void **)&d_input, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_input (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
    float *d_output = NULL;
    err = hipMalloc((void **)&d_output, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_output (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


   err = hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector h_input from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
	
   //launching process_kernel1
     int threadsPerBlock = 16;
     int blocksPerGrid = ((m*n)+threadsPerBlock-1)/threadsPerBlock; 
    
    process_kernel1<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, n, m);
	err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch process_kernel1 kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_output from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

     /*
	 // Verify that the result vectors are as expected

    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(sinf(h_input1[i]) + cosf(h_input2[i]) - h_output1[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification for h_output1 failed at element %d! value \n", i, h_input1[i]);
            exit(EXIT_FAILURE);
        }
    }
	*/

     print_array(h_output,n,m);
    

    err = hipFree(d_input);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_input (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_output);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_output (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    free(h_input);
    free(h_output);

    err = hipDeviceReset();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

   
    return 0;
}

